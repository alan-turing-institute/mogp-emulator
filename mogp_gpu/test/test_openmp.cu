
#include <hip/hip_runtime.h>
#include <iostream>
#include <omp.h>
#include <unistd.h>

#define THREAD_NUM 4

int main()
{
    
        omp_set_num_threads(THREAD_NUM); // set number of threads in "parallel" blocks
    #pragma omp parallel
    {
       // usleep(5000 * omp_get_thread_num()); // do this to avoid race condition while printing
        std::cout << "Number of available threads: " << omp_get_num_threads() << std::endl;
        // each thread can also get its own number
        std::cout << "Current thread number: " << omp_get_num_threads() << std::endl;
        std::cout << "Hello, World!" << std::endl;
    }
    
    return 0;
    
}