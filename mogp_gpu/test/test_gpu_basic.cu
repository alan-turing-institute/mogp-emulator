#include <iostream>

#include <vector>
#include <algorithm>
#include <string>
#include <sstream>
#include <assert.h>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/copy.h>

#include "../src/densegp_gpu.hpp"

#define WARP_SIZE 32
#define FULL_MASK 0xffffffff

typedef double REAL;

void test_device_vector_copy()
{
    const size_t N=5;
    // std::vector<REAL> a{1.0, 2.0, 3.0, 4.0, 5.0};
    const double a[] = {1.0, 2.0, 3.0, 4.0, 5.0};
    thrust::device_vector<REAL> a_d(N);

    std::vector<REAL> result(N);
    thrust::device_vector<REAL> result_d(N);

    thrust::copy(a, a + N, a_d.begin());

    thrust::fill(result_d.begin(), result_d.end(), 0.0);
    thrust::transform(a_d.begin(), a_d.end(), a_d.begin(), result_d.begin(), thrust::plus<float>());

    thrust::copy(result_d.begin(), result_d.end(), result.begin());

    for (int i=0; i<N; i++)
        std::cout << result[i] << " ";
    std::cout << "\n";
}



void test_sum_log_diag()
{

  size_t n=3;
  std::vector<REAL> x{1.0, 2.0, 3.0, 4., 5., 6., 7., 8., 9.};
  thrust::device_vector<REAL> x_d(x);
  // determine the size of the buffer for hipcub::device reduce


  // buffer for hipcub::DeviceReduce::Sum
  thrust::device_vector<REAL> sum_buffer_d;

    // size of sum_buffer_d
  size_t sum_buffer_size_bytes;
  // The following call determines the size of the hipcub::DeviceReduce::Sum workspace:
  // sum_buffer_size_bytes is 0 before this call, and the size of sum_buffer_d afterwards.
  // The end iterators are supplied but are not used.
  hipcub::DeviceReduce::Sum(dev_ptr(sum_buffer_d), sum_buffer_size_bytes,
			 sum_buffer_d.end(), sum_buffer_d.end(), n);
  sum_buffer_d.resize(sum_buffer_size_bytes);

  double result;
  thrust::device_vector<double> result_d(1);
  // call sum_log_diag
  sum_log_diag(n, dev_ptr(x_d), dev_ptr(result_d), dev_ptr(sum_buffer_d), sum_buffer_size_bytes);
  thrust::copy(result_d.begin(), result_d.end(), &result);

  std::cout<<"Result of sum_log_diag is "<<result<<std::endl;
}

void test_trace()
{

  size_t n=3;
  std::vector<REAL> x{1.0, 2.0, 3.0, 4., 5., 6., 7., 8., 9.};
  thrust::device_vector<REAL> x_d(x);
  // determine the size of the buffer for hipcub::device reduce

  // buffer for hipcub::DeviceReduce::Sum
  thrust::device_vector<REAL> sum_buffer_d;

    // size of sum_buffer_d
  size_t sum_buffer_size_bytes;
  // The following call determines the size of the hipcub::DeviceReduce::Sum workspace:
  // sum_buffer_size_bytes is 0 before this call, and the size of sum_buffer_d afterwards.
  // The end iterators are supplied but are not used.
  hipcub::DeviceReduce::Sum(dev_ptr(sum_buffer_d), sum_buffer_size_bytes,
			 sum_buffer_d.end(), sum_buffer_d.end(), n);
  sum_buffer_d.resize(sum_buffer_size_bytes);

  double result;
  thrust::device_vector<double> result_d(1);
  // call trace
  trace(n, dev_ptr(x_d), dev_ptr(result_d), dev_ptr(sum_buffer_d), sum_buffer_size_bytes);
  thrust::copy(result_d.begin(), result_d.end(), &result);

  std::cout<<"Result of trace is "<<result<<std::endl;
}


int main(void)
{
    test_device_vector_copy();
    test_sum_log_diag();
    test_trace();
    return 0;
}
